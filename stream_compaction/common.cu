#include "hip/hip_runtime.h"
#include "common.h"

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

void cpyHostToDevice(const int *hst, int *dev, size_t array_size) {
	hipMemcpy(	/*destination*/ dev,
				/*source*/ hst,
				/*size in bytes to copy*/ array_size,
				/*hipMemcpy type*/ hipMemcpyHostToDevice);

	checkCUDAError("Error copying memory from host to device");	
}

/**
*/
void cpyDeviceToHost(int *hst,const int *dev, size_t array_size) {
	hipMemcpy(	/*destination*/ hst,
				/*source*/ dev,
				/*size in bytes to copy*/ array_size,
				/*hipMemcpy type*/ hipMemcpyDeviceToHost);

	checkCUDAError("Error copying memory from device to host");
}

/**
*/
void cpyDeviceToDevice(const int *src, int *dest, size_t array_size) {
	hipMemcpy(	/*destination*/ dest,
				/*source*/ src,
				/*size in bytes to copy*/ array_size,
				/*hipMemcpy type*/ hipMemcpyDeviceToDevice);

	checkCUDAError("Error copying memory from device to device");
}


namespace StreamCompaction {
namespace Common {

/**
 * Maps an array to an array of 0s and 1s for stream compaction. Elements
 * which map to 0 will be removed, and elements which map to 1 will be kept.
 */
__global__ void kernMapToBoolean(int n, int *bools, const int *idata) {
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;
	
	if(index < n) {
		bools[index] = (idata[index] != 0);	
	}
}

/**
 * Performs scatter on an array. That is, for each element in idata,
 * if bools[idx] == 1, it copies idata[idx] to odata[indices[idx]].
 */
__global__ void kernScatter(int n, int *odata,
        const int *idata, const int *bools, const int *indices) {
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	if(index < n && bools[index]) {
		odata[indices[index]] = idata[index];
	}
}

}
}
